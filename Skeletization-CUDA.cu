#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#ifdef TIME
  #define COMM 1
#elif NOTIME 
  #define COMM 0
#endif

#define MASK_WIDTH 5
#define TILE_WIDTH 32
#define GPU 1
#define COMMENT "skeletization_GPU"
#define RGB_COMPONENT_COLOR 255


typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

typedef struct {
    int x, y;
} Par;

double time_total;
double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


static PPMImage *readPPM(const char *filename) {
    char buff[16];
    PPMImage *img;
    FILE *fp;
    int c, rgb_comp_color;
    fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    if (!fgets(buff, sizeof(buff), fp)) {
        perror(filename);
        exit(1);
    }

    if (buff[0] != 'P' || buff[1] != '6') {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    img = (PPMImage *) malloc(sizeof(PPMImage));
    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    c = getc(fp);
    while (c == '#') {
        while (getc(fp) != '\n')
            ;
        c = getc(fp);
    }

    ungetc(c, fp);
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
                filename);
        exit(1);
    }

    if (rgb_comp_color != RGB_COMPONENT_COLOR) {
        fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;
    img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img) {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}

void writePPM(PPMImage *img) {

    fprintf(stdout, "P6\n");
    fprintf(stdout, "# %s\n", COMMENT);
    fprintf(stdout, "%d %d\n", img->x, img->y);
    fprintf(stdout, "%d\n", RGB_COMPONENT_COLOR);

    fwrite(img->data, 3 * img->x, img->y, stdout);
    fclose(stdout);
}

void RGBtoGrayScaleImage(PPMImage *image, int *GrayScale) {

	int i;
	int cols;

	int n = image->y * image->x;

	cols = image->x;

	for (i = 0; i < n; i++) {
		GrayScale[(i/cols)*cols+(i%cols)]=(int)(0.2126*image->data[i].red+0.7152*image->data[i].green+0.0722 *image->data[i].blue);
	}
}

void Histogramify(int *GrayScale, int *histogram, int rows, int cols)
{
    int i,j;
    for(i=0; i<256; i++) histogram[i]=0;

    for(i=0; i<rows; i++)
        for(j=0; j<cols; j++)
            histogram[GrayScale[i*cols + j]]+=1;
}

int Otsu(int * histogram, int size)
{
    int i,total=size;
    float sum=0;
    for(i=0; i< 256; i++) sum+=i*histogram[i];
    
    float sumB=0;
    int wB=0;
    int wF=0;
    
    float varMax=0;
    int threshold=0;
    
    for(i=0; i<256; i++)
    {
        wB+=histogram[i];
        if(wB==0) continue;
        
        wF=total-wB;
        if(wF==0) break;
        
        sumB+=(float)(i*histogram[i]);
        float mB=sumB/wB;
        float mF=(sum-sumB)/wF;

        float varBetween=(float)wB*(float)wF*(mB-mF)*(mB-mF);
        
        if(varBetween>varMax)
        {
            varMax=varBetween;
            threshold=i;
        }
    }
    return threshold;
}   

void Neighbours(int x, int y, int **GrayScale, int *neighbours)
{
    int i,total=0;
    int X_index[8]={-1,-1,0,1,1,1,0,-1};
    int Y_index[8]={0,1,1,1,0,-1,-1,-1};
    for(i=0; i<8; i++)
    {
        neighbours[i]=GrayScale[x+X_index[i]][y+Y_index[i]];
        total+=neighbours[i]; 
    }
    neighbours[8]=total;
}

int transitions(int *neighbours)
{ 
    int i,ans=0;
    for(i=0; i<7; i++)
        if(neighbours[i]==0 && neighbours[i+1]==1) ans+=1;
    if(neighbours[7]==0 && neighbours[0]==1) ans+=1;
    return ans;
}

__global__ void Update(int *GrayScale_, int *d_changing1, int linhas, int colunas)
{
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int index = fil * colunas + col;

	if (fil>0 && col>0 && fil < linhas-1 && col < colunas-1)
	{
		if(d_changing1[index]==1)
			GrayScale_[index]=0;
	}
}
__global__ void Condition(int *GrayScale_, int *d_changing1, int *cont, int linhas, int colunas, int flag)
{
	int X_index[8]={-1,-1,0,1,1,1,0,-1};
	int Y_index[8]={0,1,1,1,0,-1,-1,-1};
	int neighbours[9]={0,0,0,0,0,0,0,0,0};
	int i,j,total=0;
	int ans=0;
	int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int index = fil * colunas + col;
	if (fil>0 && col>0 && fil < linhas-1 && col < colunas-1)
	{
		d_changing1[index]=0;
		for(i=0; i<8; i++)
		{
		    neighbours[i]=GrayScale_[(fil+X_index[i])*colunas + (col+Y_index[i])];
		    total+=neighbours[i]; 
		}
		
		neighbours[8]=total;
		for(j=0; j<7; j++)
		{
	        if(neighbours[j]==0 && neighbours[j+1]==1) 
				ans=ans+1;
		}

	    if(neighbours[7]==0 && neighbours[0]==1) 
			ans=ans+1;
		if(flag!=1)
		{
			if(GrayScale_[fil*colunas + col]==1 && neighbours[8]>=2 && neighbours[8]<=6 
				&& ans==1 && neighbours[0]*neighbours[2]*neighbours[4]==0 
				&& neighbours[2]*neighbours[4]*neighbours[6]==0)
			{
				d_changing1[index]=1;
				cont[flag]=1;
			}
		}
		else
		{
			if(GrayScale_[fil*colunas + col]==1 && neighbours[8]>=2 && neighbours[8]<=6 
				&& ans==1 && neighbours[0]*neighbours[2]*neighbours[6]==0 
				&& neighbours[0]*neighbours[4]*neighbours[6]==0)
			{
				d_changing1[index]=1;
				cont[flag]=1;
			}
		}
	}
}

void zhangsuen_CUDA(int *GrayScale, int rows, int cols)
{
	double t_start, t_end;

	//	execution time-CUDA
	double create_buffer,offload_send,kernel,offload_receive;
	create_buffer=0;
	offload_send=0;
	kernel=offload_receive=0;
	time_total=0;

    int *changing1=(int*) malloc(rows * cols *sizeof(int));
	int *d_changing1=(int*) malloc(rows * cols *sizeof(int));
	int *d_cont=(int*) malloc(2*sizeof(int));
	int *cont=(int*) malloc(2*sizeof(int));
	int *d_GrayScale;

	int sizePixel=(int) (rows * cols * sizeof(int));
	int sizeC=(int) (rows * cols *sizeof(int)); 
	int size=(int) (2*sizeof(int));

	t_start = rtclock();
	hipMalloc(&d_changing1, sizeC);
	hipMalloc(&d_GrayScale, sizePixel);
	hipMalloc(&d_cont,size);
	t_end = rtclock();
	create_buffer=create_buffer+(t_end - t_start);
	
	//  define number of blocks and threads
	int par1=(int)ceil((float)cols / TILE_WIDTH);
	int par2=(int) ceil((float)rows / TILE_WIDTH);
		
	dim3 dimGrid(par1,par2,1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	*changing1=0;
	cont[0]=cont[1]=1;
	while(cont[0]>0 || cont[1]>0)
    {  
		cont[0]=0;
		cont[1]=0;
		/*=========================== First Condition ================================*/		
		//  copy memory to the GPU here
		t_start = rtclock();
		hipMemcpy(d_GrayScale,GrayScale,sizePixel,hipMemcpyHostToDevice);
		hipMemcpy(d_changing1,changing1,sizeC,hipMemcpyHostToDevice);
		hipMemcpy(d_cont,cont,size,hipMemcpyHostToDevice);
		t_end = rtclock();
		offload_send=offload_send+(t_end - t_start);
		
		//  launch the GPU Kernel here
		t_start = rtclock();
		Condition<<<dimGrid,dimBlock>>>(d_GrayScale,d_changing1,d_cont,rows,cols,0);
		hipDeviceSynchronize();
		t_end = rtclock();
		kernel=kernel+(t_end - t_start);

		//  copy the GPU memory back to the CPU here   
		t_start = rtclock();
		hipMemcpy(changing1, d_changing1, sizeC, hipMemcpyDeviceToHost);
		hipMemcpy(cont, d_cont,size, hipMemcpyDeviceToHost);
		t_end = rtclock();
		offload_receive=offload_receive+(t_end - t_start);

		/*============ First Update ============*/
		//  copy memory to the GPU here
		t_start = rtclock();
		hipMemcpy(d_GrayScale,GrayScale,sizePixel,hipMemcpyHostToDevice);
		hipMemcpy(d_changing1,changing1,sizeC,hipMemcpyHostToDevice);
		t_end = rtclock();
		offload_send=offload_send+(t_end - t_start);

		//  launch the GPU Kernel here
		t_start = rtclock();
		Update<<<dimGrid,dimBlock>>>(d_GrayScale,d_changing1,rows,cols);
		hipDeviceSynchronize();
		t_end = rtclock();
		kernel=kernel+(t_end - t_start);

		//  copy the GPU memory back to the CPU here   
		t_start = rtclock();
		hipMemcpy(GrayScale, d_GrayScale, sizePixel, hipMemcpyDeviceToHost);
		t_end = rtclock();
		offload_receive=offload_receive+(t_end - t_start);
		
		cont[1]=0;
		/*========================== Second Condition ===========================*/		
		//  copy memory to the GPU here
		t_start = rtclock();
		hipMemcpy(d_GrayScale,GrayScale,sizePixel,hipMemcpyHostToDevice);
		hipMemcpy(d_changing1,changing1,sizeC,hipMemcpyHostToDevice);
		hipMemcpy(d_cont,cont,size,hipMemcpyHostToDevice);
		t_end = rtclock();
		offload_send=offload_send+(t_end - t_start);
		
		//  launch the GPU Kernel here
		t_start = rtclock();
		Condition<<<dimGrid,dimBlock>>>(d_GrayScale,d_changing1,d_cont,rows,cols,1);
		hipDeviceSynchronize();
		t_end = rtclock();
		kernel=kernel+(t_end - t_start);

		//  copy the GPU memory back to the CPU here   
		t_start = rtclock();
		hipMemcpy(changing1, d_changing1, sizeC, hipMemcpyDeviceToHost);
		hipMemcpy(cont, d_cont,size, hipMemcpyDeviceToHost);
		t_end = rtclock();
		offload_receive=offload_receive+(t_end - t_start);

		/*============ First Update ============*/
		//  copy memory to the GPU here
		t_start = rtclock();
		hipMemcpy(d_GrayScale,GrayScale,sizePixel,hipMemcpyHostToDevice);
		hipMemcpy(d_changing1,changing1,sizeC,hipMemcpyHostToDevice);
		t_end = rtclock();
		offload_send=offload_send+(t_end - t_start);

		//  launch the GPU Kernel here
		t_start = rtclock();
		Update<<<dimGrid,dimBlock>>>(d_GrayScale,d_changing1,rows,cols);
		hipDeviceSynchronize();
		t_end = rtclock();
		kernel=kernel+(t_end - t_start);

		//  copy the GPU memory back to the CPU here   
		t_start = rtclock();
		hipMemcpy(GrayScale, d_GrayScale, sizePixel, hipMemcpyDeviceToHost);
		t_end = rtclock();
		offload_receive=offload_receive+(t_end - t_start);
    }
	time_total=create_buffer+offload_send+kernel+offload_receive;
	if(COMM==1)
	{
		printf("Parallel time : %0.6lfs \n", time_total);
		printf("	create buffer : %0.6lfs \n", create_buffer);
		printf("	offload send : %0.6lfs \n",offload_send);
		printf("	kernel : %0.6lfs \n", kernel);
		printf("	offload receive : %0.6lfs \n", offload_receive);
	}
	hipFree(d_GrayScale);
	hipFree(d_changing1);
}

void zhangsuen(int *GrayScale, int rows, int cols)
{
    int *neighbours=(int *) malloc(9*sizeof (int));
    int *changing1=(int*) malloc(rows * cols *sizeof(int));
	int *GrayScale_ = (int*) malloc(rows*cols * sizeof(int));
    int i,total,cont1=1;
    int j,k,cont2=1;
	int ans=0;
    int X_index[8]={-1,-1,0,1,1,1,0,-1};
    int Y_index[8]={0,1,1,1,0,-1,-1,-1};
    while(cont1>0 || cont2>0)
    {
        cont1=0;
        cont2=0;
        for(i=1; i<rows-1; i++)
        {   
            for(j=1; j<cols-1; j++)
            {       
					total=0;
					ans=0;
					changing1[i*cols+j]=0;
					for(k=0; k<8; k++)
					{
						neighbours[k]=GrayScale[(i+X_index[k])*cols + (j+Y_index[k])];
						total+=neighbours[k]; 
					}
					neighbours[8]=total;

		            for(k=0; k<7; k++)
					{
						if(neighbours[k]==0 && neighbours[k+1]==1) 
							ans=ans+1;
					}

					if(neighbours[7]==0 && neighbours[0]==1) 
						ans=ans+1;

					if(GrayScale[i*cols+j]==1 && neighbours[8]>=2 && neighbours[8]<=6 && ans==1 && neighbours[0]*neighbours[2]*neighbours[4]==0 && neighbours[2]*neighbours[4]*neighbours[6]==0)
					{
						changing1[i*cols+j]=1;
					}
            }
        }

        for(i=1; i<rows-1; i++)
		{
		    for(j=1; j<cols-1; j++)
			{
				if(changing1[i*cols +j]==1){
					cont1=cont1+1;
		        	GrayScale[i*cols+j]=0;
				}
			}
		}
        
        for(i=1; i<rows-1; i++)
        {
            for(j=1; j<cols-1; j++)
            {
					total=0;
					ans=0;
					changing1[i*cols+j]=0;
					for(k=0; k<8; k++)
					{
						neighbours[k]=GrayScale[(i+X_index[k])*cols + (j+Y_index[k])];
						total+=neighbours[k]; 
					}
					neighbours[8]=total;

		            for(k=0; k<7; k++)
					{
						if(neighbours[k]==0 && neighbours[k+1]==1) 
							ans=ans+1;
					}

					if(neighbours[7]==0 && neighbours[0]==1) 
						ans=ans+1;

					if(GrayScale[i*cols+j]==1 && neighbours[8]>=2 && neighbours[8]<=6 && ans==1 && neighbours[0]*neighbours[2]*neighbours[6]==0 && neighbours[0]*neighbours[4]*neighbours[6]==0)
					{
						changing1[i*cols+j]=1;
					}
            }
        }
        for(i=1; i<rows-1; i++)
		{
		    for(j=1; j<cols-1; j++)
			{
				if(changing1[i*cols +j]==1){
					cont2=cont2+1;
		        	GrayScale[i*cols+j]=0;
				}
			}
		}
	
    }
}

int main(int argc, char *argv[]) {

    if( argc != 2 ) {
        printf("Too many or no one arguments supplied.\n");
    }

	double t_start, t_end;
    int rows, cols,i,j;
	if(COMM==1)
		printf("====== %s ======\n",argv[1]);
	// read image
	PPMImage *image = readPPM(argv[1]);
	cols = image->x;
	rows = image->y;

	int *histogram= (int *) malloc(sizeof (int) *256);
	int *GrayScale = (int*) malloc(image->x * image->y * sizeof(int));
	int *GrayScale2 = (int*) malloc(image->x * image->y * sizeof(int));

	//	convert image from RGG to Gray scale
	RGBtoGrayScaleImage(image,GrayScale);	

	//	histogram of the gray scale image
	Histogramify(GrayScale,histogram, rows, cols);

	int var_otsu=Otsu(histogram,rows*cols);
		
	//	convert image from gray scale to binary image
	for(i=0; i<rows; i++)
		   for(j=0; j<cols; j++)
		       if(GrayScale[i*cols + j]<var_otsu) GrayScale[i*cols + j]=GrayScale2[i*cols + j]=1;
		       else GrayScale[i*cols + j]=GrayScale2[i*cols + j]=0;
	//	skeletonize binary image
	t_start = rtclock();
	zhangsuen(GrayScale,rows,cols);
	t_end = rtclock();

	if(COMM==1)
		fprintf(stdout, "\nSerial time: %0.6lfs\n", t_end - t_start); 

	zhangsuen_CUDA(GrayScale2,rows,cols);

	if(COMM==1)
		fprintf(stdout, "Speedup: %0.3lf\n\n",(double)(t_end - t_start)/(double)time_total); 

	for(i=0; i<rows; i++)
	{
		for(j=0; j<cols; j++)
		{
		   if(GrayScale[i*cols + j]==1)
		   {
		       image->data[i*cols+j].red=255;
		       image->data[i*cols+j].green=255;
		       image->data[i*cols+j].blue=255;
		   } 
		   else
		   {
		       image->data[i*cols+j].red=0;
		       image->data[i*cols+j].green=0;
		       image->data[i*cols+j].blue=0;
		   }       
		}
	}
	if(COMM==0)
	{
		writePPM(image);
		free(image);
	}

}
